#include "hip/hip_runtime.h"
//
// Created by russoul on 12.04.18.
//

#include "hip/hip_vector_types.h"
#include "cuda_noise.cuh"
#include <>
#include <cstdlib>
#include <iostream>
#include <zconf.h>
#include <vector>
#include <sys/time.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

struct vec3f{
    float array[3];
};


__host__ __device__ __forceinline__ vec3f make_vec3f(float x, float y, float z){
    vec3f ret;
    ret.array[0] = x;
    ret.array[1] = y;
    ret.array[2] = z;

    return ret;
}

__host__ __device__ __forceinline__ vec3f operator+(vec3f a, vec3f b){
    vec3f ret;

    ret.array[0] = a.array[0] + b.array[0];
    ret.array[1] = a.array[1] + b.array[1];
    ret.array[2] = a.array[2] + b.array[2];

    return ret;
}

__host__ __device__ __forceinline__ vec3f operator-(vec3f a, vec3f b){
    vec3f ret;

    ret.array[0] = a.array[0] - b.array[0];
    ret.array[1] = a.array[1] - b.array[1];
    ret.array[2] = a.array[2] - b.array[2];

    return ret;
}


__host__ __device__ __forceinline__ vec3f operator*(vec3f a, float k){
    vec3f ret;

    ret.array[0] = a.array[0] * k;
    ret.array[1] = a.array[1] * k;
    ret.array[2] = a.array[2] * k;

    return ret;
}

__host__ __device__ __forceinline__ vec3f operator/(vec3f a, float k){
    vec3f ret;

    ret.array[0] = a.array[0] / k;
    ret.array[1] = a.array[1] / k;
    ret.array[2] = a.array[2] / k;

    return ret;
}

__host__ __device__ __forceinline__ float dot(vec3f a, vec3f b){
    return a.array[0] * b.array[0] + a.array[1] * b.array[1] + a.array[2] * b.array[2];
}

__host__ __device__ __forceinline__ float norm(vec3f a){
    return sqrtf(dot(a,a));
}

__host__ __device__ __forceinline__ vec3f normalize(vec3f a){
    return a / norm(a);
}

__host__ __device__ __forceinline__ vec3f fromFloat3(float3 a){
    return make_vec3f(a.x, a.y, a.z);
}

__host__ __device__ __forceinline__ float3 toFloat3(vec3f a){
    return make_float3(a.array[0], a.array[1], a.array[2]);
}


std::string dump_float3(float3 v){
    return "(x = " + std::to_string(v.x) + ", y = " + std::to_string(v.y) + ", z = " + std::to_string(v.z) + ")";
}

struct Line3{
    float3 start;
    float3 end;
};

//=============== uniform voxel storage ==================
struct HermiteData{
    float3 intersection;
    float3 normal;
};

struct UniformVoxelStorage{
    uint cellCount;
    float* grid;
    HermiteData** edgeInfo;
};
//========================================================


__constant__ int specialTable1[768];

__constant__ uint specialTable2[12];

__constant__ float3 cornerPoints[8];

__constant__ uint2 edgePairs[12];


inline __device__ __host__ uint indexDensity(uint cellCount, uint x, uint y, uint z){
    return z * (cellCount + 2) * (cellCount + 2) + y * (cellCount + 2) + x;
}

inline __device__ __host__ uint indexCell(uint cellCount, uint x, uint y, uint z){
    return z * (cellCount + 1) * (cellCount + 1) + y * (cellCount + 1) + x;
}

inline __device__ float denSphere(float3 offset, float rad, float3 p){
    return dot(p - offset, p - offset) - rad * rad;
}

inline __device__ float octaveNoise(size_t octaves, float persistence, float x, float y, float z, int seed){
    float total = 0.0F;
    float frequency = 1.0F;
    float amplitude = 1.0F;
    float maxValue = 0.0;

    float k = powf(2.0, octaves - 1);

    for (int i = 0; i < octaves; ++i) {
        total += cudaNoise::simplexNoise(make_float3(x * frequency / k, y * frequency / k, z * frequency / k),1, seed);
        maxValue += amplitude;
        amplitude *= persistence;
        frequency *= 2.0;
    }

    return total / maxValue;
}

inline __device__ float denFn(float3 p, int seed, float ymin, float extent){
    auto den = (cudaNoise::repeaterPerlin(make_float3(p.x, 0, p.z), 0.5F, seed, 8, 0.85, 0.95) + 1.01 )/2 * 2 * extent * 0.7F;
    return p.y - ymin - den;
   /* auto den = denSphere(make_float3(2,2,2), 2, p);
    return den;*/

}


inline __device__ void loadDensity(uint x, uint y, uint z, float3 offset, float a, UniformVoxelStorage storage, int seed, float ymin, float extent){
    auto p = offset + make_float3(x * a, y * a, z * a);
    auto den = denFn(p, seed, ymin, extent);
    //printf("%f for px=%f py=%f pz=%f x=%u y=%u z=%u i=%u a=%f size=%u\n", den, p.x, p.y, p.z, x,y,z, indexDensity(storage.cellCount, x,y,z),a,storage.cellCount);


    storage.grid[indexDensity(storage.cellCount, x,y,z)] = den;

    //printf("%f for px=%f py=%f pz=%f x=%u y=%u z=%u i=%u a=%f size=%u\n", storage.grid[indexDensity(storage.cellCount, x,y,z)], p.x, p.y, p.z, x,y,z, indexDensity(storage.cellCount, x,y,z),a,storage.cellCount);
}

__global__ void kernelLoadDensity(float3 offset, float a, UniformVoxelStorage storage, int seed, float ymin, float extent){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    //printf("kernel bx=%d, tx=%d\n", blockIdx.x, threadIdx.x);

    uint x = i % (storage.cellCount + 2);
    uint y = (i / (storage.cellCount + 2)) % (storage.cellCount + 2);
    uint z = (i / (storage.cellCount + 2) / (storage.cellCount + 2)) % (storage.cellCount + 2);

    loadDensity(x,y,z, offset, a, storage, seed, ymin, extent);
}

__device__ float3 sampleSurfaceIntersection(Line3 line, uint n, int seed, float ymin, float extent){
    auto ext = line.end - line.start;

    auto norm = length(ext);
    auto dir = ext / norm;

    auto center = line.start + ext * 0.5F;
    auto curExt = norm * 0.25F;

    for (int i = 0; i < n; ++i) {
        auto point1 = center - dir * curExt;
        auto point2 = center + dir * curExt;
        auto den1 = fabsf(denFn(point1, seed, ymin, extent));
        auto den2 = fabsf(denFn(point2, seed, ymin, extent));

        if(den1 <= den2){
            center = point1;
        }else{
            center = point2;
        }
    }

    return center;
}

__device__ float3 calculateNormal(float3 point, float eps, int seed, float ymin, float extent){
    float d = denFn(point, seed, ymin, extent);
    return normalize(make_float3(denFn(make_float3(point.x + eps, point.y, point.z), seed, ymin, extent) - d,
                                 denFn(make_float3(point.x, point.y + eps, point.z), seed, ymin, extent) - d,
                                 denFn(make_float3(point.x, point.y, point.z + eps), seed, ymin, extent) - d
    ));
}

__global__ void markCell(uint indexOffset, UniformVoxelStorage storage, bool* marks){
    int i = blockIdx.x * blockDim.x + threadIdx.x + indexOffset;

    uint x = i % (storage.cellCount + 1);
    uint y = (i / (storage.cellCount + 1)) % (storage.cellCount + 1);
    uint z = (i / (storage.cellCount + 1) / (storage.cellCount + 1)) % (storage.cellCount + 1);


    uint config = 0;

    if(storage.grid[indexDensity(storage.cellCount, x,y,z)] < 0.0){
        config |= 1;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y,z)] < 0.0){
        config |= 2;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y,z+1)] < 0.0){
        config |= 4;
    }
    if(storage.grid[indexDensity(storage.cellCount, x,y,z+1)] < 0.0){
        config |= 8;
    }

    if(storage.grid[indexDensity(storage.cellCount, x,y+1,z)] < 0.0){
        config |= 16;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y+1,z)] < 0.0){
        config |= 32;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y+1,z+1)] < 0.0){
        config |= 64;
    }
    if(storage.grid[indexDensity(storage.cellCount, x,y+1,z+1)] < 0.0){
        config |= 128;
    }



    if(specialTable1[3 * config] != -2){
        marks[i] = 1;
    }else{
        marks[i] = 0;
    }
}


bool operator <(float3 a, float3 b){
    if(a.x < b.x){
        return true;
    }else if(a.x > b.x){
        return false;
    }else{
        if(a.y < b.y){
            return true;
        }else if(a.y > b.y){
            return false;
        }else{
            if(a.z < b.z){
                return true;
            }else if(a.z >= b.z){
                return false;
            }
        }
    }
}

//TODO needed ?
//given two points in space (line): find the minimum one of two them (component-wise in lexical order) and then find floating point number alpha = sqrt((middle - min)*(middle - min)/(max - min)*(max - min))
//used for unique alpha identification
float makeAlphaComponent(float3 end1, float3 end2, float3 middle){
    if(end1 < end2){//min = end1, max = end2
        return sqrtf(dot(middle - end1,middle - end1) / dot(end2 - end1, end2 - end1));
    }else{
        return sqrtf(dot(middle - end2,middle - end2) / dot(end2 - end1, end2 - end1));
    }
}

float3 makeMiddleFromAlphaComponent(float3 end1, float3 end2, float alpha){
    if(end1 < end2){
        return end1 + normalize(end2 - end1) * alpha;
    }else{
        return end2 + normalize(end2 - end1) * alpha;
    }
}

__global__ void loadCell(uint indexOffset, float3 offset, float a, uint acc, UniformVoxelStorage storage, int seed, uint* marked, uint markedLen, HermiteData* data, float ymin, float extent){
    uint i_ = blockIdx.x * blockDim.x + threadIdx.x + indexOffset;

    if(i_ >= markedLen) return;


    uint i = marked[i_];

    uint x = i % (storage.cellCount + 1);
    uint y = (i / (storage.cellCount + 1)) % (storage.cellCount + 1);
    uint z = (i / (storage.cellCount + 1) / (storage.cellCount + 1)) % (storage.cellCount + 1);

    auto cellMin = offset + make_float3(x * a, y * a, z * a);

    uint config = 0;

    if(storage.grid[indexDensity(storage.cellCount, x,y,z)] < 0.0){
        config |= 1;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y,z)] < 0.0){
        config |= 2;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y,z+1)] < 0.0){
        config |= 4;
    }
    if(storage.grid[indexDensity(storage.cellCount, x,y,z+1)] < 0.0){
        config |= 8;
    }

    if(storage.grid[indexDensity(storage.cellCount, x,y+1,z)] < 0.0){
        config |= 16;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y+1,z)] < 0.0){
        config |= 32;
    }
    if(storage.grid[indexDensity(storage.cellCount, x+1,y+1,z+1)] < 0.0){
        config |= 64;
    }
    if(storage.grid[indexDensity(storage.cellCount, x,y+1,z+1)] < 0.0){
        config |= 128;
    }



    int* entry = specialTable1 + 3 * config;


    //printf("%u %u %d\n", i, config,  *entry);

    //if(*entry != -2){ this is guaranteed by markCell's
    int curEntry = entry[0];

    while(curEntry != -2){
        auto corners = edgePairs[curEntry];
        Line3 edge = {cellMin + cornerPoints[corners.x] * a, cellMin + cornerPoints[corners.y] * a};
        auto intersection = sampleSurfaceIntersection(edge, acc, seed, ymin, extent);
        auto normal = calculateNormal(intersection, a / 1024.0F, seed, ymin, extent);


        data[3 * i_ + specialTable2[curEntry]] = {intersection, normal};

        curEntry = *(++entry);
    }

    storage.edgeInfo[indexCell(storage.cellCount, x,y,z)] = data + 3 * i_;
    //}
}

extern "C" void testVec3f(float3 a){
    printf("x=%f, y=%f, z=%f\n", a.x, a.y, a.z);
    printf("sizeof float3 = %d\n", sizeof(float3));

}


unsigned long long timeMs(){
    struct timeval tv;

    gettimeofday(&tv, NULL);

    unsigned long long millisecondsSinceEpoch =
            (unsigned long long)(tv.tv_sec) * 1000 +
            (unsigned long long)(tv.tv_usec) / 1000;

    return millisecondsSinceEpoch;
}


extern "C" void sampleGPU(float3 offset, float a, uint acc, UniformVoxelStorage* storage){
    auto size = storage->cellCount;

    printf("info: ox=%f oy=%f oz=%f a=%f\n size=%d", offset.x, offset.y, offset.z, a, size);

    std::cout << "start" << std::endl;
    std::flush(std::cout);

    int seed = static_cast<int>(time(NULL));

    printf("seed=%i\n", seed);

    auto t7 = timeMs();
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024); //set 1G available memory

    float extent = size * a / 2;
    float ymin = offset.y;

    float* grid_d;
    HermiteData** edgeInfo_d;
    bool* marks_d; //TODO use bitmaps for more efficient storage of bools
    bool* marks = static_cast<bool *>(malloc(sizeof(bool) * (size + 1) * (size + 1) * (size + 1)));
    uint* marked_d;
    HermiteData* data_d;
    gpuErrchk(hipMalloc(&grid_d, sizeof(float) * (size + 2) * (size + 2) * (size + 2)));
    gpuErrchk(hipMalloc(&edgeInfo_d, sizeof(HermiteData*) * (size + 1)*(size + 1)*(size + 1)));
    gpuErrchk(hipMalloc(&marks_d, sizeof(bool) * (size + 1)*(size + 1)*(size + 1)));
    hipMemset(edgeInfo_d, 0, sizeof(HermiteData*) * (size + 1)*(size + 1)*(size + 1));
    auto t8 = timeMs();
    std::cout << "memory preallocation took " << (t8 - t7) << " ms" << std::endl;



    UniformVoxelStorage storage_d = {size, grid_d, edgeInfo_d};


    //std::cout << "before density" << std::endl;
    auto t1 = timeMs();
    kernelLoadDensity<<<(size+2)*(size+2),(size+2)>>>(offset, a, storage_d, seed, ymin, extent);
    gpuErrchk(hipDeviceSynchronize());
    auto t2 = timeMs();

    std::cout << "density loading (GPU part) took " << (t2 - t1) << " ms" << std::endl;



    //std::cout << "after density" << std::endl;
    //std::flush(std::cout);

    auto t3 = timeMs();
    markCell<<<(size+1)*(size+1),(size+1)>>>(0, storage_d, marks_d);
    gpuErrchk(hipDeviceSynchronize());
    auto t4 = timeMs();
    std::cout << "cell marking (GPU part) took " << (t4 - t3) << " ms" << std::endl;
    gpuErrchk(hipMemcpy(marks, marks_d, sizeof(bool) * (size+1)*(size+1)*(size+1), hipMemcpyDeviceToHost));

    std::vector<uint> indices;

    auto t10 = timeMs();
    for (uint l = 0; l < (size+1)*(size+1)*(size+1); ++l) {
        if(marks[l]){
            indices.push_back(l);
        }
    }
    auto t11 = timeMs();
    std::cout << "cell marking (CPU part) took " << (t11 - t10) << " ms" << std::endl;

    auto t12 = timeMs();
    gpuErrchk(hipMalloc(&marked_d, sizeof(uint) * indices.size()));
    gpuErrchk(hipMemcpy(marked_d, &indices[0], sizeof(uint) * indices.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipFree(marks_d));
    gpuErrchk(hipMalloc(&data_d, sizeof(HermiteData) * 3 * indices.size()));
    auto t13 = timeMs();
    std::cout << "mid allocation took " << (t13 - t12) << " ms" << std::endl;


    std::cout << "index count = " << indices.size() << std::endl;
    std::flush(std::cout);

    uint blockSize = 256;

    uint invokations = indices.size() / blockSize + 1;

    auto t5 = timeMs();
    loadCell<<<invokations,256>>>(0, offset, a, (uint)(log2f(acc) + 1), storage_d, seed, marked_d, indices.size(), data_d, ymin, extent);
    gpuErrchk(hipDeviceSynchronize());
    auto t6 = timeMs();
    std::cout << "hermite data loading (GPU part) took " << (t6 - t5) << " ms" << std::endl;


    auto t15 = timeMs();
    gpuErrchk(hipFree(marked_d));
    gpuErrchk(hipMemcpy(storage->grid, storage_d.grid, sizeof(float) * (size + 2) * (size + 2) * (size + 2), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(grid_d));
    gpuErrchk(hipMemcpy(storage->edgeInfo, storage_d.edgeInfo, sizeof(HermiteData*) * (size + 1) * (size + 1) * (size + 1), hipMemcpyDeviceToHost));

    HermiteData* data = static_cast<HermiteData *>(malloc(sizeof(HermiteData) * 3 * indices.size())); //TODO pass back and free
    gpuErrchk(hipMemcpy(data, data_d, sizeof(HermiteData) * 3 * indices.size(), hipMemcpyDeviceToHost));


    #pragma omp parallel for
    for (int j = 0; j < (size + 1)*(size + 1)*(size + 1); ++j) {
        HermiteData* ptr_d = storage->edgeInfo[j];

        if(ptr_d){
            storage->edgeInfo[j] = (ptrdiff_t)(ptr_d - data_d) + data;
        }
    }

    gpuErrchk(hipFree(edgeInfo_d));
    gpuErrchk(hipFree(data_d));
    auto t16 = timeMs();
    std::cout << "post CPU took " << (t16 - t15) << " ms" << std::endl;

}


extern "C" void setConstantMem(){
    int specialTable1_local[256][3] = {


            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {-2, -2, -2},
            {0, 3, 8},
            {0, -2, -2},
            {3, 8, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {3, -2, -2},
            {0, 8, -2},
            {0, 3, -2},
            {8, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {8, -2, -2},
            {0, 3, -2},
            {0, 8, -2},
            {3, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},
            {3, 8, -2},
            {0, -2, -2},
            {0, 3, 8},
            {-2, -2, -2},

    };


    float3 cornerPoints_local[8] = {
            make_float3(0.0f, 0.0f, 0.0f),
            make_float3(1.0f, 0.0f, 0.0f), //clockwise starting from zero y min
            make_float3(1.0f, 0.0f, 1.0f),
            make_float3(0.0f, 0.0f, 1.0f),


            make_float3(0.0f, 1.0f, 0.0f),
            make_float3(1.0f, 1.0f, 0.0f), //y max
            make_float3(1.0f, 1.0f, 1.0f),
            make_float3(0.0f, 1.0f, 1.0f)
    };

    uint2 edgePairs_local[12] = {
            make_uint2(0,1),
            make_uint2(1,2),
            make_uint2(3,2),
            make_uint2(0,3),

            make_uint2(4,5),
            make_uint2(5,6),
            make_uint2(7,6),
            make_uint2(4,7),

            make_uint2(4,0),
            make_uint2(1,5),
            make_uint2(2,6),
            make_uint2(3,7)
    };

    for (int i = 0; i < 256; ++i) {
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(specialTable1), specialTable1_local[i], sizeof(int) * 3, sizeof(int) * 3 * i));
    }


    uint specialTable2_local[12] = {0,1,0,1,0,1,0,1,2,2,2,2};

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(specialTable2), specialTable2_local, sizeof(uint) * 12));


    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cornerPoints), cornerPoints_local, sizeof(float3) * 8));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(edgePairs), edgePairs_local, sizeof(uint2) * 12));
}